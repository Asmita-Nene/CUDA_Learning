#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void addNums(int a, int b, int* res){
  *res = a + b;
  printf("Added the numbers in this function inside the GPU\n");
}

int main(){
  int a = 10;
  int b = 15;
  int res = 0;

  int*  device_res_ptr;

  hipMalloc(&device_res_ptr, sizeof(int));

  addNums<<<1, 1>>>(a, b, device_res_ptr);
  hipDeviceSynchronize();

  hipMemcpy(&res, device_res_ptr, sizeof(int), hipMemcpyDeviceToHost);
  cout<<"\nResult is: "<<res<<endl;

  hipFree(device_res_ptr);

  return 0;
}